#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include "heat-2d_kernel.hu"
/*
 * Discretized 2D heat equation stencil with non periodic boundary conditions
 * Adapted from Pochoir test bench
 *
 * Irshad Pananilath: irshad@csa.iisc.ernet.in
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <math.h>
#ifdef USE_LIKWID
#include<likwid.h>
#endif


/*
 * N is the number of points
 * T is the number of timesteps
 */
#ifdef HAS_DECLS
#include "decls.h"
#else
#define N 4000L
//#define T 1000L
#endif

#define NUM_FP_OPS 10

/* Define our arrays */
double A[2][N+2][N+2];
double total=0; double sum_err_sqr=0;
int chtotal=0;
int timeval_subtract(struct timeval *result, struct timeval *x, struct timeval *y) {
    if (x->tv_usec < y->tv_usec) {
        int nsec = (y->tv_usec - x->tv_usec) / 1000000 + 1;

        y->tv_usec -= 1000000 * nsec;
        y->tv_sec += nsec;
    }

    if (x->tv_usec - y->tv_usec > 1000000) {
        int nsec = (x->tv_usec - y->tv_usec) / 1000000;

        y->tv_usec += 1000000 * nsec;
        y->tv_sec -= nsec;
    }

    result->tv_sec = x->tv_sec - y->tv_sec;
    result->tv_usec = x->tv_usec - y->tv_usec;

    return x->tv_sec < y->tv_sec;
}

int main(int argc, char * argv[]) {
    long int i, j;
    const int BASE = 1024;

    // for timekeeping
    struct timeval start, end, result;
    double tdiff = 0.0;
    
    int T;

    printf("Please enter number of timesteps = \n");
    scanf("%d", &T);

    printf("Number of points = %ld\t|Number of timesteps = %ld\t", N*N, T);

    /* Initialization */
    srand(42); // seed with a constant value to verify results

    for (i = 0; i <= N+1; i++) {
        for (j = 0; j <= N+1; j++) {
            A[0][i][j] = 1.0 * (rand() % BASE);
        }
    }

#ifdef USE_LIKWID
#pragma omp parallel
{
LIKWID_MARKER_START("Compute_omp");
}
#endif

    if (T >= 1)
      {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

        double *dev_A;
        
        cudaCheckReturn(hipMalloc((void **) &dev_A, (2) * (4002) * (4002) * sizeof(double)));
        
        cudaCheckReturn(hipMemcpy(dev_A, A, (2) * (4002) * (4002) * sizeof(double), hipMemcpyHostToDevice));

#ifdef TIME
        gettimeofday(&start, 0);
#endif
        
        #define floord(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
        for (int c0 = 0; c0 < T; c0 += 1)
          {
            dim3 k0_dimBlock(16, 32);
            dim3 k0_dimGrid(126, 126);
            kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_A, T, c0);
            cudaCheckKernel();
            cudaCheckReturn(hipDeviceSynchronize());
          }

#ifdef TIME
        gettimeofday(&end, 0);

        timeval_subtract(&result, &end, &start);
        tdiff += (double)(result.tv_sec + result.tv_usec * 1.0e-6);
#endif
          
        cudaCheckReturn(hipMemcpy(A, dev_A, (2) * (4002) * (4002) * sizeof(double), hipMemcpyDeviceToHost));
        
        cudaCheckReturn(hipFree(dev_A));
      }

#ifdef TIME
    printf("|Time taken =  %7.5lfms\t", tdiff * 1.0e3);
    printf("|MFLOPS =  %f\t", ((((double)NUM_FP_OPS * N *N *  T) / tdiff) / 1000000L));
#endif

#ifdef USE_LIKWID
#pragma omp parallel
{
LIKWID_MARKER_STOP("Compute_omp");
}
#endif


#ifdef VERIFY
    for (i = 1; i < N+1; i++) {
        for (j = 1; j < N+1; j++) {
            total+= A[T%2][i][j] ;
        }
    }
    printf("|sum: %e\t", total);
    for (i = 1; i < N+1; i++) {
        for (j = 1; j < N+1; j++) {
            sum_err_sqr += (A[T%2][i][j] - (total/N))*(A[T%2][i][j] - (total/N));
        }
    }
    printf("|rms(A) = %7.2f\t", sqrt(sum_err_sqr));
    for (i = 1; i < N+1; i++) {
        for (j = 1; j < N+1; j++) {
            chtotal += ((char *)A[T%2][i])[j];
        }
    }
    printf("|sum(rep(A)) = %d\n", chtotal);
#endif
    return 0;
}

// icc -O3 -fp-model precise heat_1d_np.c -o op-heat-1d-np -lm
// /* @ begin PrimeTile (num_tiling_levels=1; first_depth=1; last_depth=-1; boundary_tiling_level=-1;) @*/
// /* @ begin PrimeRegTile (scalar_replacement=0; T1t3=8; T1t4=8; ) @*/
// /* @ end @*/
